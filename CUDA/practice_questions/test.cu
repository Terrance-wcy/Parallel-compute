
#include<stdio.h>
#include<stdlib.h>
#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;
 
#define M 33
#define N 33
#define I 33
#define block_size 32
 
#define size_a M*I*sizeof(float)
#define size_b I*N*sizeof(float)
#define size_c M*N*sizeof(float)
 
__global__ void matrix_kernel(float *d_a,float *d_b,float *d_c,int wA,int wB);//wA,wB是A,B矩阵的宽度，也就是列数
void showResult(float *c,int h,int Kw,int w);
void init(float *a,int h,int Kw,int w,int value);//Kw为扩充后矩阵宽度，而w为实际矩阵宽度
//考虑将矩阵补齐，就是行列都变成32*32块的整数倍,函数功能是补齐并初始化,还要分配空间
float* supply(int h,int w,int value);//参数为行，宽，就可以得到扩充后的大小,value为需要初始化的值
int main()
{
 
	float *h_a,*h_b,*h_c;//声明主机指针
	float *d_a,*d_b,*d_c;//声明设备指针
 
	//在考虑补齐的时候,实际行数列数都发生了变化,记住最后的矩阵大小为row_c*col_c,但是实际矩阵大小为M*N即可
	int row_a=32*((M+32-1)/32);//补齐后a的行列，但是从[32*M/32,M)为初始化数值，而[M,row_a)则应初始化为零
	int col_a=32*((I+32-1)/32);//[I,col_a)初始化为零
	cout<<"a矩阵扩充后的行列分别为："<<row_a<<" "<<col_a<<" "<<(M+31)/32<<" "<<32*((M+32-1)/32)<<endl;
	int row_b=col_a;//[I,row_b)行，初始化为0
	int col_b=32*((N+32-1)/32);//[N,col_b）列初始化为0
	int row_c=row_a;
	int col_c=col_b;
	cout<<"c矩阵扩充后的行列分别为："<<row_c<<" "<<col_c<<" "<<(M+31)/32<<" "<<32*((M+32-1)/32)<<endl;
	cout<<"b矩阵扩充后的行列分别为："<<row_b<<" "<<col_b<<" "<<(M+31)/32<<" "<<32*((M+32-1)/32)<<endl;
	int size_a_a=row_a*col_a*sizeof(float);//补齐之后的矩阵所占用空间
	int size_b_b=row_b*col_b*sizeof(float);
	int size_c_c=row_c*col_c*sizeof(float);
	cout<<"a,b,c矩阵扩充后所占用空间："<<size_a_a<<" "<<size_b_b<<" "<<size_c_c<<endl;
 
	h_a=supply(row_a,col_a,1);//分配空间并初始化
	//显示扩充后a矩阵
	showResult(h_a,row_a,col_a,col_a);
	h_b=supply(row_b,col_b,2);
	showResult(h_b,row_b,col_b,col_b);
	h_c=supply(row_c,col_c,0);
	showResult(h_c,row_c,col_c,col_c);
	/*
	h_a=(float*)malloc(size_a);init(h_a,M,I,1);//主机上分配空间并初始化
	h_b=(float*)malloc(size_b);init(h_b,I,N,2);showResult(h_a,M,I);cout<<"a矩阵输出完成！"<<endl;showResult(h_b,I,N);cout<<"b矩阵输出完成"<<endl;
	h_c=(float*)malloc(size_c);memset(h_c,0,size_c);
	*/
	if(h_a==NULL||h_b==NULL||h_c==NULL)
	{
		fprintf(stderr,"分配主机内存失败!\n");
		exit(EXIT_FAILURE);
	}
 
	//设备上分配空间并检查
	hipError_t err=hipSuccess;//设定成功标志码
	err=hipMalloc((void**)&d_a,size_a_a);
	hipMalloc((void**)&d_b,size_b_b);
	hipMalloc((void**)&d_c,size_c_c);hipMemset(d_c,0,size_c_c);
	/*
	err=cudaMalloc((void**)&d_a,size_a);
	cudaMalloc((void**)&d_b,size_b);
	cudaMalloc((void**)&d_c,size_c);cudaMemset(d_c,0,size_c);//只对c初始化为0
	*/
	if(err!=hipSuccess)
	{
		fprintf(stderr,"分配设备内存a失败！\n");
		exit(EXIT_FAILURE);
	}
	//加入时间计时代码
	hipEvent_t start,stop;//声明事件
	hipEventCreate(&start);//创建事件
	hipEventCreate(&stop);
	hipEventRecord(start,0);//sample使用NULL流，应该和这里一样反正都是默认流的意思
	//#################GPU工作部分######################矩阵大小变化了，后面这些东西都要变
	hipMemcpy(d_a,h_a,size_a_a,hipMemcpyHostToDevice);//主机向device拷贝数据，对于错误的检查，就不加了，太麻烦
	hipMemcpy(d_b,h_b,size_b_b,hipMemcpyHostToDevice);
	dim3 dimsA(col_a,row_a);//这里始终是第一个参数为列，第二个参数为行
	dim3 dimsB(col_b,row_b);
	//dim3 grids((N+block_size-1)/block_size,(M+block_size-1)/block_size);//按照我的补齐思路其实也不用这样写，因为始终是齐的
														//这里一定要记住了，每次都犯错误，对于cuda中的dim3写法，三个元素（x,y,z）对应的就是列，行，高，
														//所以说在这里写个grid（32,64）表示32列，64行
	dim3 grids(col_c/block_size,row_c/block_size);
	dim3 threads(block_size,block_size);
	//调用kernel函数计算内积
	matrix_kernel<<<grids,threads>>>(d_a,d_b,d_c,dimsA.x,dimsB.x);
	err=hipMemcpy(h_c,d_c,size_c_c,hipMemcpyDeviceToHost);//将结果数据拷贝回主机内存
	if(err!=hipSuccess)
	{
		fprintf(stderr,"拷贝数据回主机失败！\n");
		exit(EXIT_FAILURE);
	}
	hipEventRecord(stop,0);//将记录结束时间放入GPU工作队列中
	hipEventSynchronize(stop);//CPU等待GPU完成所有工作后，再继续执行
	float totalTime=0.0f;//定义存储时间的变量，使用float型
	hipEventElapsedTime(&totalTime,start,stop);
	//至此所有工作全部完成，剩下就是输出和释放了
	printf("GPU计算时间为：%f\n",totalTime);
	showResult(h_c,M,col_c,N);//显示输出结果
	cout<<M<<'\t'<<N<<endl;
	showResult(h_c,row_c,col_c,col_c);//显示全部的大矩阵
	cout<<row_c<<'\t'<<col_c<<endl;
	//释放资源
	free(h_a);free(h_b);free(h_c);
	hipFree(d_a);hipFree(d_b);hipFree(d_c);
	hipEventDestroy(start);hipEventDestroy(stop);	
	return 0;
}
//这里核函数的线程调度为block（32,32） grid（20，10），即grid是20列，10行 
__global__ void matrix_kernel(float *d_a,float *d_b,float *d_c,int wA,int wB)
{
	//block index
	int bx=blockIdx.x,by=blockIdx.y;
	//thread index
	int tx=threadIdx.x,ty=threadIdx.y;
	//这里只是考虑的是对齐的情况，对于每一个block而言，先得到block的初始指针位置aBegin实际上表示的是一种位置的偏移量，指针加上偏移量就得到值，本身不使用指针更方便
	int aBegin=by*block_size*wA;//其实就是by*32*I，I就是A的宽度，这就很显然了
	int aEnd=aBegin+wA-1;//其实aBegin和aEnd就是一行的首尾位置
	int aStep=block_size;//这里注意a的步幅是在for循环中的跨度，因为每次只是载入一个块，所以整行要分多个块，就是这块之间的距离
	int bBegin=bx*block_size;//画图看的话，很清楚。分清楚，对于不同的block有不同的aBegin和bBegin，与for循环中跨度区别
	int bStep=block_size*wB;
	float Csub=0;//每个线程都会计算出一个行列的相乘值出来，放在Csub中
	for(int a=aBegin,b=bBegin;a<aEnd;a+=aStep,b+=bStep)
	{
		__shared__ float as[block_size][block_size];
		__shared__ float bs[block_size][block_size];
		as[ty][tx]=d_a[a+ty*wA+tx];//这里其实本质上这样看：a+tx其实就是在多少列上，这样看也行，怎么看都行，或者看为a为起始位置，加上ty*wA其实就是再下移ty行的位置，在加上tx就可以了
		bs[ty][tx]=d_b[b+ty*wB+tx];//bs与as取值的规律本质上是一样的。仅仅只是矩阵不同，而导致宽度不同而已
		__syncthreads();//保证数据都载入shared
		//第二个for循环计算乘累加
		for(int i=0;i<block_size;i++)
		{
			Csub+=as[ty][i]*bs[i][tx];
		}
		__syncthreads();//做到这里就表示一个分块的小矩阵计算完成，再进入下一个分块的小矩阵
	}
	//当所有循环做完后，每个线程都得到一个Csub值，写入d_c相应位置即可
	int c=wB*by*block_size+bx*block_size;//c在这里其实表示的是每个block的起始位置，要得到每个thread的在grid中的位置，还要加上在block中的相对位置
	d_c[c+ty*wB+tx]=Csub;
}
//初始化数据函数,我采取的策略是对矩阵进行两次赋值，每次赋值的行列不相同，
void init(float *a,int h,int Kw,int w,int value)//w始终是扩充后矩阵的宽，也就是列数，h为高，也就是行数,
{
	for(int i=0;i<h;i++)
	{
		for(int j=0;j<w;j++)
			a[i*Kw+j]=value;
	}
}
//显示结果函数
void showResult(float *c,int h,int Kw,int w)//h高度就是行，w宽度就是列
{
	for(int i=0;i<h;i++)
	{
		for(int j=0;j<w;j++)
		{
			cout<<c[i*Kw+j]<<'\t';
		}
		cout<<endl;
	}
}
//补齐矩阵函数
float* supply(int h,int w,int value)
{
	float *p=(float*)malloc(h*w*sizeof(float));
	init(p,h,w,w,0);//先将全矩阵初始化为0,
	init(p,M,w,I,value);//再讲实际需要运算的部分初始化为value，这里需要扩充矩阵的宽度，和需要实际赋值的矩阵宽度
	return p;
}
